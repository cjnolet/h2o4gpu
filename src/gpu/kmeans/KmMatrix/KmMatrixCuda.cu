/*!
 * Copyright 2018 H2O.ai, Inc.
 * License   Apache License Version 2.0 (see LICENSE for details)
 */

#include <stdexcept>
#include <iostream>
#include <memory>

#include <thrust/device_vector.h>

#include "KmMatrixCuda.cuh"
#include "KmMatrix.hpp"
#include "backend.hpp"

namespace H2O4GPU {
namespace KMeans {

template <typename T>
CudaKmMatrixImpl<T>::CudaKmMatrixImpl(KmMatrix<T> * _par) :
    KmMatrixImpl<T>(_par){}

template <typename T>
CudaKmMatrixImpl<T>::CudaKmMatrixImpl(const thrust::host_vector<T>& _h_vec,
                                      KmMatrix<T>* _par)
    : on_device_(false), KmMatrixImpl<T>(_par) {
  h_vector_.resize(_h_vec.size());
  thrust::copy(_h_vec.begin(), _h_vec.end(), h_vector_.begin());
}

template <typename T>
CudaKmMatrixImpl<T>::CudaKmMatrixImpl(size_t _size, KmMatrix<T> * _par) :
    KmMatrixImpl<T>(_par) {
  if (_size == 0) return;

  d_vector_.resize(_size);
  on_device_ = true;
}

template <typename T>
CudaKmMatrixImpl<T>::CudaKmMatrixImpl(
    KmMatrix<T>& _other, size_t _start, size_t _size, size_t _stride,
    KmMatrix<T> * _par) :
    KmMatrixImpl<T>(_par) {
  assert (_size > 0);

  if (_size == 0)
    return;

  T* raw_ptr;

  assert (raw_ptr != nullptr && raw_ptr != NULL);

  std::cerr << "Warning: Copying data from " << _other.name()
            << "." << std::endl;
  if (_other.on_device()) {
    raw_ptr = _other.dev_ptr();
    thrust::device_ptr<T> ptr (raw_ptr);
    ptr += _start;
    d_vector_.resize(_size);
    on_device_ = true;
    thrust::copy(ptr, ptr + _size, d_vector_.begin());
  } else {
    raw_ptr = _other.host_ptr();
    raw_ptr += _start;
    h_vector_.resize(_size);
    on_device_ = false;
    thrust::copy(raw_ptr, raw_ptr + _size, h_vector_.begin());
  }
}

template <typename T>
CudaKmMatrixImpl<T>::~CudaKmMatrixImpl() {}

template <typename T>
void CudaKmMatrixImpl<T>::set_interface(KmMatrix<T>* _par) {
  KmMatrixImpl<T>::matrix_ = _par;
}

template <typename T>
T* CudaKmMatrixImpl<T>::host_ptr() {
  device_to_host();
  return thrust::raw_pointer_cast(h_vector_.data());
}

template <typename T>
T* CudaKmMatrixImpl<T>::dev_ptr() {
  host_to_device();
  T* ptr = thrust::raw_pointer_cast(d_vector_.data());
  return ptr;
}

template <typename T>
void CudaKmMatrixImpl<T>::host_to_device() {
  if (on_device_)
    return;
  d_vector_.resize(h_vector_.size());
  thrust::copy(h_vector_.begin(), h_vector_.end(), d_vector_.begin());
  on_device_ = true;
}

template <typename T>
void CudaKmMatrixImpl<T>::device_to_host() {
  if (!on_device_)
    return;
  h_vector_.resize(d_vector_.size());
  thrust::copy(d_vector_.begin(), d_vector_.end(), h_vector_.begin());
  on_device_ = false;
}

template <typename T>
bool CudaKmMatrixImpl<T>::on_device() const {
  return on_device_;
}

template <typename T>
size_t CudaKmMatrixImpl<T>::size() const {
  if (on_device_) {
    return d_vector_.size();
  } else {
    return h_vector_.size();
  }
}

template <typename T>
bool CudaKmMatrixImpl<T>::equal(KmMatrix<T>& _rhs) {
  T* rhs_raw_ptr = _rhs.dev_ptr();
  host_to_device();
  thrust::device_ptr<T> rhs_ptr (rhs_raw_ptr);
  // FIXME, Is it floating compatible?
  bool res = thrust::equal(d_vector_.begin(), d_vector_.end(),
                           rhs_ptr);
  return res;
}

template <typename T>
KmMatrix<T> CudaKmMatrixImpl<T>::stack(KmMatrix<T>& _second,
                                       KmMatrixDim _dim) {
  if (_dim == KmMatrixDim::ROW) {
    if (KmMatrixImpl<T>::matrix_->cols() != _second.cols()) {
      M_ERROR("Columns of first is not equal to second.");
    }
    host_to_device();

    T * sec_raw_ptr = _second.dev_ptr();
    thrust::device_ptr<T> self_ptr = d_vector_.data();

    thrust::device_ptr<T> sec_ptr (sec_raw_ptr);

    KmMatrix<T> res (KmMatrixImpl<T>::matrix_->rows() + _second.rows(),
                     KmMatrixImpl<T>::matrix_->cols());

    T * res_raw_ptr = res.dev_ptr();
    thrust::device_ptr<T> res_ptr (res_raw_ptr);

    thrust::copy(self_ptr, self_ptr + size(), res_ptr);
    res_ptr = thrust::device_ptr<T>(res_raw_ptr) + size();
    thrust::copy(sec_ptr, sec_ptr + _second.size(), res_ptr);

    return res;
  } else {
    // FIXME
    M_ERROR("Not implemented.");
  }
}


#define INSTANTIATE(T)                                                  \
  /* Standard con(de)structors*/                                        \
  template CudaKmMatrixImpl<T>::CudaKmMatrixImpl(                       \
      KmMatrix<T>& _other, size_t _start, size_t _size, size_t _stride, \
      KmMatrix<T> * _par);                                              \
  template CudaKmMatrixImpl<T>::CudaKmMatrixImpl(                       \
      const thrust::host_vector<T>& _h_vec, KmMatrix<T>* _par);         \
  template CudaKmMatrixImpl<T>::CudaKmMatrixImpl(KmMatrix<T> * _par);   \
  template CudaKmMatrixImpl<T>::CudaKmMatrixImpl(size_t _size,          \
                                                 KmMatrix<T> * _par);   \
  template CudaKmMatrixImpl<T>::~CudaKmMatrixImpl();                    \
  template void CudaKmMatrixImpl<T>::set_interface(KmMatrix<T>* _par);  \
  /* Member functions */                                                \
  template bool CudaKmMatrixImpl<T>::on_device() const;                 \
  template void CudaKmMatrixImpl<T>::device_to_host();                  \
  template void CudaKmMatrixImpl<T>::host_to_device();                  \
  template T* CudaKmMatrixImpl<T>::dev_ptr();                           \
  template T* CudaKmMatrixImpl<T>::host_ptr();                          \
  template size_t CudaKmMatrixImpl<T>::size() const;                    \
  template bool CudaKmMatrixImpl<T>::equal(KmMatrix<T>& _rhs);  \
  template KmMatrix<T> CudaKmMatrixImpl<T>::stack(KmMatrix<T>& _second, \
                                                  KmMatrixDim _dim);

INSTANTIATE(float)
INSTANTIATE(double)
INSTANTIATE(int)

#undef INSTANTIATE
}  // namespace H204GPU
}  // namespace Array
